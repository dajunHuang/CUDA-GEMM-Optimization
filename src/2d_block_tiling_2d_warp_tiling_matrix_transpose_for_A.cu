#include "hip/hip_runtime.h"
#include "cuda_gemm.hpp"
#include "cuda_gemm_utils.cuh"
#include "cuda_gemm_utils.hpp"
#include <hip/hip_cooperative_groups.h>
#include "stdio.h"
#include <cstddef>
#include <cstdio>
#include <iostream>

// template <typename T, size_t BLOCK_TILE_SIZE_X, size_t BLOCK_TILE_SIZE_Y,
//           size_t NUM_THREADS_PER_BLOCK>
// __device__ void set_global_memory_to_zero(T * C_thread_block_tile,
//     size_t ldsc, size_t warp_linear_idx, size_t thread_linear_idx_in_warp)
// {
//     constexpr unsigned int NUM_EACH_THREAD_LOAD_C{(BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_Y) /
//         NUM_THREADS_PER_BLOCK};
//     constexpr unsigned int NUM_LOAD_THREADS_PER_ROW_C{BLOCK_TILE_SIZE_Y /  
//         NUM_EACH_THREAD_LOAD_C};
//     size_t const c_load_warp_row_idx{warp_linear_idx / NUM_LOAD_THREADS_PER_ROW_C};
//     size_t const c_load_warp_col_idx{warp_linear_idx % NUM_LOAD_THREADS_PER_ROW_C};

//     size_t const block_row{c_load_warp_row_idx * 32U + thread_linear_idx_in_warp};
//     size_t const block_col{c_load_warp_col_idx * NUM_EACH_THREAD_LOAD_C};

//     for(size_t load_time{0}; load_time < NUM_EACH_THREAD_LOAD_C; ++load_time)
//     {
//         C_thread_block_tile[block_row + (block_col + load_time) * ldsc] = 0;
//     }
// }

template <typename T, size_t BLOCK_TILE_SIZE, size_t WARP_TILE_SIZE,
          size_t NUM_THREAD_TILES_PER_WARP, size_t THREAD_TILE_SIZE>
__device__ void load_data_from_shared_memory_to_register_file(
    T const *thread_block_tile,
    T register_values[NUM_THREAD_TILES_PER_WARP][THREAD_TILE_SIZE],
    size_t warp_idx, size_t thread_idx)
{
    static_assert(BLOCK_TILE_SIZE % THREAD_TILE_SIZE == 0U);
#pragma unroll
    for (size_t thread_tile_repeat_idx{0U};
         thread_tile_repeat_idx < NUM_THREAD_TILES_PER_WARP;
         ++thread_tile_repeat_idx)
    {
        size_t const thread_block_tile_idx{
            warp_idx * WARP_TILE_SIZE + thread_tile_repeat_idx *
                (WARP_TILE_SIZE / NUM_THREAD_TILES_PER_WARP) +
                thread_idx * THREAD_TILE_SIZE};
#pragma unroll
        for (size_t thread_tile_idx{0U}; thread_tile_idx < THREAD_TILE_SIZE;
                ++thread_tile_idx)
        {
            register_values[thread_tile_repeat_idx][thread_tile_idx] = 
                thread_block_tile[thread_block_tile_idx + thread_tile_idx];
        }
    }
}

template <typename T, size_t WARP_TILE_SIZE_X, size_t WARP_TILE_SIZE_Y,
          size_t NUM_THREAD_TILES_PER_WARP_X, size_t NUM_THREAD_TILES_PER_WARP_Y,
          size_t NUM_THREADS_PER_WARP_X, size_t NUM_THREADS_PER_WARP_Y,
          size_t THREAD_TILE_SIZE_X, size_t THREAD_TILE_SIZE_Y>
__device__ void compute_thread_tile_results(
    T const A_vals[NUM_THREAD_TILES_PER_WARP_X][THREAD_TILE_SIZE_X],
    T const B_vals[NUM_THREAD_TILES_PER_WARP_Y][THREAD_TILE_SIZE_Y],
    T C_thread_results[WARP_TILE_SIZE_X][WARP_TILE_SIZE_Y],
    size_t const thread_linear_row_idx_in_warp,
    size_t const thread_linear_col_idx_in_warp)
{
#pragma unroll
    for (size_t thread_tile_repeat_row_idx{0U};
         thread_tile_repeat_row_idx < NUM_THREAD_TILES_PER_WARP_X;
         ++thread_tile_repeat_row_idx)
    {
#pragma unroll
        for (size_t thread_tile_repeat_col_idx{0U};
             thread_tile_repeat_col_idx < NUM_THREAD_TILES_PER_WARP_Y;
             ++thread_tile_repeat_col_idx)
        {
            size_t const C_thread_row_idx{
                thread_tile_repeat_row_idx * (WARP_TILE_SIZE_X /
                NUM_THREAD_TILES_PER_WARP_X) +
                thread_linear_row_idx_in_warp * THREAD_TILE_SIZE_X};
            size_t const C_thread_col_idx{
                thread_tile_repeat_col_idx * (WARP_TILE_SIZE_Y /
                NUM_THREAD_TILES_PER_WARP_Y) +
                thread_linear_col_idx_in_warp * THREAD_TILE_SIZE_Y};
#pragma unroll
            for (size_t thread_tile_x_idx{0U};
                 thread_tile_x_idx < THREAD_TILE_SIZE_X; ++thread_tile_x_idx)
            {
#pragma unroll
                for (size_t thread_tile_y_idx{0U};
                     thread_tile_y_idx < THREAD_TILE_SIZE_Y;
                     ++thread_tile_y_idx)
                {
                    // TODO transpose? Is there register bank?
                    // What's the best order for loops?
                    // Right?
                    C_thread_results[C_thread_row_idx + thread_tile_x_idx]
                        [C_thread_col_idx + thread_tile_y_idx] +=
                        A_vals[thread_tile_repeat_row_idx][thread_tile_x_idx] *
                        B_vals[thread_tile_repeat_col_idx][thread_tile_y_idx];
                }
            }
        }
    }
}

template <typename T, size_t BLOCK_TILE_SIZE_X, size_t BLOCK_TILE_SIZE_Y,
          size_t NUM_WARPS_X, size_t NUM_WARPS_Y,
          size_t WARP_TILE_SIZE_X, size_t WARP_TILE_SIZE_Y,
          size_t NUM_THREAD_TILES_PER_WARP_X, size_t NUM_THREAD_TILES_PER_WARP_Y,
          size_t NUM_THREADS_PER_WARP_X, size_t NUM_THREADS_PER_WARP_Y,
          size_t THREAD_TILE_SIZE_X, size_t THREAD_TILE_SIZE_Y>
__device__ void copy_results_from_register_to_shared_memory(
    T C_thread_results[WARP_TILE_SIZE_X][WARP_TILE_SIZE_Y],
    T *C_thread_block_tile, size_t const ldc, size_t const m, size_t const n,
    size_t const warp_row_idx, size_t const warp_col_idx,
    size_t const thread_linear_row_idx_in_warp,
    size_t const thread_linear_col_idx_in_warp)
{
#pragma unroll
    for (size_t thread_tile_repeat_row_idx{0U};
         thread_tile_repeat_row_idx < NUM_THREAD_TILES_PER_WARP_X;
         ++thread_tile_repeat_row_idx)
    {
#pragma unroll
        for (size_t thread_tile_repeat_col_idx{0U};
             thread_tile_repeat_col_idx < NUM_THREAD_TILES_PER_WARP_Y;
             ++thread_tile_repeat_col_idx)
        {
            size_t const C_thread_warp_row_idx{
                thread_tile_repeat_row_idx * (WARP_TILE_SIZE_X /
                NUM_THREAD_TILES_PER_WARP_X) +
                thread_linear_row_idx_in_warp * THREAD_TILE_SIZE_X};
            size_t const C_thread_warp_col_idx{
                thread_tile_repeat_col_idx * (WARP_TILE_SIZE_Y /
                NUM_THREAD_TILES_PER_WARP_Y) +
                thread_linear_col_idx_in_warp * THREAD_TILE_SIZE_Y};
            size_t const C_thread_block_tile_row_idx{
                warp_row_idx * WARP_TILE_SIZE_X + C_thread_warp_row_idx};
            size_t const C_thread_block_tile_col_idx{
                warp_col_idx * WARP_TILE_SIZE_Y + C_thread_warp_col_idx};
#pragma unroll
            for (size_t thread_tile_x_idx{0U};
                 thread_tile_x_idx < THREAD_TILE_SIZE_X; ++thread_tile_x_idx)
            {
#pragma unroll
                for (size_t thread_tile_y_idx{0U};
                     thread_tile_y_idx < THREAD_TILE_SIZE_Y;
                     ++thread_tile_y_idx)
                {
                    C_thread_block_tile[(C_thread_block_tile_row_idx + thread_tile_x_idx) +
                        (C_thread_block_tile_col_idx + thread_tile_y_idx) * ldc] = 
                        C_thread_results[C_thread_warp_row_idx + thread_tile_x_idx]
                        [C_thread_warp_col_idx + thread_tile_y_idx];

                }
            }
        }
    }
}

// template <typename T, size_t BLOCK_TILE_SIZE_X, size_t BLOCK_TILE_SIZE_Y,
//           size_t NUM_THREADS_PER_BLOCK>
// __device__ void copy_results_from_shared_to_global_memory(
//     size_t block_row_idx, size_t block_col_idx, T const* C_thread_block_tile,
//     size_t ldsc, T* C_work, size_t ldc, size_t warp_linear_idx,
//     size_t thread_linear_idx_in_warp)
// {
//     constexpr unsigned int NUM_EACH_THREAD_LOAD_C{(BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_Y) /
//         NUM_THREADS_PER_BLOCK};
//     constexpr unsigned int NUM_LOAD_THREADS_PER_ROW_C{BLOCK_TILE_SIZE_Y /  
//         NUM_EACH_THREAD_LOAD_C};
//     size_t const c_load_warp_row_idx{warp_linear_idx / NUM_LOAD_THREADS_PER_ROW_C};
//     size_t const c_load_warp_col_idx{warp_linear_idx % NUM_LOAD_THREADS_PER_ROW_C};

//     size_t const block_row{c_load_warp_row_idx * 32U + thread_linear_idx_in_warp};
//     size_t const block_col{c_load_warp_col_idx * NUM_EACH_THREAD_LOAD_C};
//     size_t const c_row{block_row_idx * BLOCK_TILE_SIZE_X + block_row};
//     size_t const c_col{block_col_idx * BLOCK_TILE_SIZE_Y + block_col};

//     for(size_t load_time{0}; load_time < NUM_EACH_THREAD_LOAD_C; ++load_time)
//     {
//         C_work[c_row + (c_col + load_time) * ldc] +=
//             C_thread_block_tile[block_row + (block_col + load_time) * ldsc];
//     }
// }

template <typename T, size_t BLOCK_TILE_SIZE_X,
          size_t BLOCK_TILE_SIZE_K, size_t BLOCK_TILE_SIZE_Y,
          size_t BLOCK_TILE_SKEW_SIZE_A, size_t BLOCK_TILE_SKEW_SIZE_B,
          size_t WARP_TILE_SIZE_X, size_t WARP_TILE_SIZE_Y,
          size_t NUM_THREADS_PER_WARP_X, size_t NUM_THREADS_PER_WARP_Y,
          size_t THREAD_TILE_SIZE_X, size_t THREAD_TILE_SIZE_Y>
__global__ void gemm_tall_and_skinny(size_t m, size_t n, size_t k, T const* alpha,
                                     T const* A, size_t lda, T const* B, size_t ldb,
                                     T const* beta, T* C, size_t ldc)
{
    constexpr size_t NUM_WARPS_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    constexpr size_t NUM_WARPS_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};

    constexpr unsigned int NUM_THREAD_TILES_PER_WARP_X{
        WARP_TILE_SIZE_X / (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X)};
    constexpr unsigned int NUM_THREAD_TILES_PER_WARP_Y{
        WARP_TILE_SIZE_Y / (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y)};

    // Avoid using blockDim.x * blockDim.y as the number of threads per block.
    // Because it is a runtime constant and the compiler cannot optimize the
    // loop unrolling based on that.
    // Use a compile time constant instead.
    constexpr unsigned int NUM_THREADS_PER_BLOCK{NUM_WARPS_X * NUM_WARPS_Y * 32U};

    // Cache a tile of A and B in shared memory for data reuse.
    // A is col majored, B is row majored, C is col majored.
    constexpr size_t ldsa = BLOCK_TILE_SIZE_X + BLOCK_TILE_SKEW_SIZE_A;
    __shared__ T A_thread_block_tile[BLOCK_TILE_SIZE_K * ldsa];
    constexpr size_t ldsb = BLOCK_TILE_SIZE_Y + BLOCK_TILE_SKEW_SIZE_B;
    __shared__ T B_thread_block_tile[BLOCK_TILE_SIZE_K * ldsb];
    constexpr size_t ldsc = BLOCK_TILE_SIZE_X + BLOCK_TILE_SKEW_SIZE_A;
    // TODO shared memory of C should be initialized to all 0.
    __shared__ T C_thread_block_tile[BLOCK_TILE_SIZE_Y * ldsc];

    size_t const thread_linear_idx{threadIdx.x};
    size_t const warp_linear_idx{thread_linear_idx / 32U};
    size_t const thread_linear_idx_in_warp{thread_linear_idx % 32U};

    // set_global_memory_to_zero<T, BLOCK_TILE_SIZE_X, BLOCK_TILE_SIZE_Y,
    //     NUM_THREADS_PER_BLOCK>
    //     (C_thread_block_tile, ldsc, warp_linear_idx, thread_linear_idx_in_warp);


    // A_vals is cached in the register.
    T A_vals[NUM_THREAD_TILES_PER_WARP_X][THREAD_TILE_SIZE_X] = {
        static_cast<T>(0)};
    // B_vals is cached in the register.
    T B_vals[NUM_THREAD_TILES_PER_WARP_Y][THREAD_TILE_SIZE_Y] = {
        static_cast<T>(0)};

    constexpr unsigned int NUM_EACH_THREAD_LOAD_A{
        (BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_K) / NUM_THREADS_PER_BLOCK};
    constexpr unsigned int NUM_LOAD_THREADS_PER_ROW_A{
        BLOCK_TILE_SIZE_K / NUM_EACH_THREAD_LOAD_A};
    size_t const a_load_warp_row_idx{warp_linear_idx / NUM_LOAD_THREADS_PER_ROW_A};
    size_t const a_load_warp_col_idx{warp_linear_idx % NUM_LOAD_THREADS_PER_ROW_A};

    // if(blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("threadIdx: %ld a_load_warp_row_idx: %ld a_load_warp_col_idx: %ld NUM_EACH_THREAD_LOAD_A: %d NUM_LOAD_THREADS_PER_ROW_A: %d\n",
    //         thread_linear_idx, a_load_warp_row_idx, a_load_warp_col_idx, NUM_EACH_THREAD_LOAD_A, NUM_LOAD_THREADS_PER_ROW_A);
    // }

    #pragma unroll
    for(size_t load_time{0}; load_time < NUM_EACH_THREAD_LOAD_A; ++load_time)
    {
        A_thread_block_tile[(a_load_warp_row_idx * 32U + thread_linear_idx_in_warp) +
                            (a_load_warp_col_idx * NUM_EACH_THREAD_LOAD_A + load_time) * ldsa] =
            A[(blockIdx.x * BLOCK_TILE_SIZE_X + a_load_warp_row_idx * 32U + thread_linear_idx_in_warp) +
              (blockIdx.y * BLOCK_TILE_SIZE_K + a_load_warp_col_idx * NUM_EACH_THREAD_LOAD_A + load_time) * lda];
    }

    constexpr unsigned int NUM_EACH_THREAD_LOAD_B{(BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_Y) / NUM_THREADS_PER_BLOCK};
    constexpr unsigned int NUM_LOAD_THREADS_PER_ROW_B{BLOCK_TILE_SIZE_Y / NUM_EACH_THREAD_LOAD_B};
    size_t const b_load_warp_row_idx{warp_linear_idx / NUM_LOAD_THREADS_PER_ROW_B};
    size_t const b_load_warp_col_idx{warp_linear_idx % NUM_LOAD_THREADS_PER_ROW_B};

    // if(blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("threadIdx: %ld b_load_warp_row_idx: %ld b_load_warp_col_idx: %ld NUM_EACH_THREAD_LOAD_B: %d NUM_LOAD_THREADS_PER_ROW_B: %d\n",
    //         thread_linear_idx, b_load_warp_row_idx, b_load_warp_col_idx, NUM_EACH_THREAD_LOAD_B, NUM_LOAD_THREADS_PER_ROW_B);
    // }

    #pragma unroll
    for(size_t load_time{0}; load_time < NUM_EACH_THREAD_LOAD_B; ++load_time)
    {
        B_thread_block_tile[(b_load_warp_row_idx * 32U + thread_linear_idx_in_warp) * ldsb +
                            (b_load_warp_col_idx * NUM_EACH_THREAD_LOAD_B + load_time)] =
            B[(blockIdx.y * BLOCK_TILE_SIZE_K + b_load_warp_row_idx * 32U + thread_linear_idx_in_warp) +
              (b_load_warp_col_idx * NUM_EACH_THREAD_LOAD_B + load_time) * ldb];
    }
    __syncthreads();

    // Check shared memory load
    //
    // if(thread_linear_idx == 0)
    // {
    //     // printf("BLOCK_TILE_SIZE_X: %ld BLOCK_TILE_SIZE_K: %ld\n", BLOCK_TILE_SIZE_X, BLOCK_TILE_SIZE_K);
    //     for(size_t x_i{0}; x_i < BLOCK_TILE_SIZE_X; ++x_i)
    //     {
    //         for(size_t k_i{0}; k_i < BLOCK_TILE_SIZE_K; ++k_i)
    //         {
    //             if(A[(blockIdx.x * BLOCK_TILE_SIZE_X + x_i) +
    //             (blockIdx.y * BLOCK_TILE_SIZE_K + k_i) * lda]
    //             != A_thread_block_tile[x_i + k_i * ldsa])
    //                 printf("A share meme copy err, A_thread_block_tile[%ld][%ld]\n", x_i, k_i);
    //         }
    //     }
    //     for(size_t k_i{0}; k_i < BLOCK_TILE_SIZE_K; ++k_i)
    //     {
    //         for(size_t y_i{0}; y_i < BLOCK_TILE_SIZE_Y; ++y_i)
    //         {
    //             if(B[(blockIdx.y * BLOCK_TILE_SIZE_K + k_i) + (y_i) * ldb] !=
    //                B_thread_block_tile[k_i * ldsb + y_i])
    //                 printf("A share meme copy err, B_thread_block_tile[%ld][%ld]\n", k_i, y_i);
    //         }
    //     }
    // }

    size_t const warp_row_idx{warp_linear_idx / NUM_WARPS_Y};
    size_t const warp_col_idx{warp_linear_idx % NUM_WARPS_Y};
    size_t const thread_linear_row_idx_in_warp{thread_linear_idx_in_warp /
        NUM_THREADS_PER_WARP_Y};
    size_t const thread_linear_col_idx_in_warp{thread_linear_idx_in_warp %
        NUM_THREADS_PER_WARP_Y};

    T C_thread_results[WARP_TILE_SIZE_X][WARP_TILE_SIZE_Y] = {
        static_cast<T>(0)};

    #pragma unroll
    for (size_t k_i{0U}; k_i < BLOCK_TILE_SIZE_K; ++k_i)
    {
        // Load data from shared memory to register file for A.
        load_data_from_shared_memory_to_register_file<T, BLOCK_TILE_SIZE_X,
            WARP_TILE_SIZE_X, NUM_THREAD_TILES_PER_WARP_X, 
            THREAD_TILE_SIZE_X>
            (A_thread_block_tile + k_i * ldsa, A_vals, warp_row_idx, thread_linear_row_idx_in_warp);

        // Load data from shared memory to register file for B.
        load_data_from_shared_memory_to_register_file<T, BLOCK_TILE_SIZE_Y,
            WARP_TILE_SIZE_Y, NUM_THREAD_TILES_PER_WARP_Y,
            THREAD_TILE_SIZE_Y>
            (B_thread_block_tile + k_i * ldsb, B_vals, warp_col_idx, thread_linear_col_idx_in_warp);

        // Check register load
        //
        // for (size_t thread_tile_repeat_idx{0U};
        //     thread_tile_repeat_idx < NUM_THREAD_TILES_PER_WARP_X;
        //     ++thread_tile_repeat_idx)
        // {
        //     size_t const thread_block_tile_idx{
        //         warp_row_idx * WARP_TILE_SIZE_X + thread_tile_repeat_idx *
        //             (WARP_TILE_SIZE_X / NUM_THREAD_TILES_PER_WARP_X) +
        //             thread_linear_row_idx_in_warp * THREAD_TILE_SIZE_X};
        //     for (size_t thread_tile_idx{0U}; thread_tile_idx < THREAD_TILE_SIZE_X;
        //             ++thread_tile_idx)
        //     {
        //         if(A_vals[thread_tile_repeat_idx][thread_tile_idx] != 
        //             A_thread_block_tile[k_i * ldsa + thread_block_tile_idx + thread_tile_idx])
        //         {
        //             printf("A reg load error\n");   
        //         }

        //     }
        // }
        // for (size_t thread_tile_repeat_idx{0U};
        //     thread_tile_repeat_idx < NUM_THREAD_TILES_PER_WARP_Y;
        //     ++thread_tile_repeat_idx)
        // {
        //     size_t const thread_block_tile_idx{
        //         warp_col_idx * WARP_TILE_SIZE_Y + thread_tile_repeat_idx *
        //             (WARP_TILE_SIZE_Y / NUM_THREAD_TILES_PER_WARP_Y) +
        //             thread_linear_col_idx_in_warp * THREAD_TILE_SIZE_Y};
        //     for (size_t thread_tile_idx{0U}; thread_tile_idx < THREAD_TILE_SIZE_Y;
        //             ++thread_tile_idx)
        //     {
        //         if(B_vals[thread_tile_repeat_idx][thread_tile_idx] != 
        //             B_thread_block_tile[k_i * ldsb + thread_block_tile_idx + thread_tile_idx])
        //         {
        //             printf("B reg load error\n");   
        //         }

        //     }
        // }

        // Compute NUM_THREAD_TILES_PER_WARP_X * NUM_THREAD_TILES_PER_WARP_Y
        // outer products.
        compute_thread_tile_results<T, WARP_TILE_SIZE_X, WARP_TILE_SIZE_Y,
                                    NUM_THREAD_TILES_PER_WARP_X,
                                    NUM_THREAD_TILES_PER_WARP_Y,
                                    NUM_THREADS_PER_WARP_X,
                                    NUM_THREADS_PER_WARP_Y,
                                    THREAD_TILE_SIZE_X, THREAD_TILE_SIZE_Y>
                                    (A_vals, B_vals, C_thread_results,
                                        thread_linear_row_idx_in_warp,
                                        thread_linear_col_idx_in_warp);   
    }

    __syncthreads();

    // Add the results of threads to shared_memory.
    copy_results_from_register_to_shared_memory<T,
        BLOCK_TILE_SIZE_X, BLOCK_TILE_SIZE_Y,
        NUM_WARPS_X, NUM_WARPS_Y,
        WARP_TILE_SIZE_X, WARP_TILE_SIZE_Y,
        NUM_THREAD_TILES_PER_WARP_X, NUM_THREAD_TILES_PER_WARP_Y,
        NUM_THREADS_PER_WARP_X, NUM_THREADS_PER_WARP_Y,
        THREAD_TILE_SIZE_X, THREAD_TILE_SIZE_Y>
        (C_thread_results, C_thread_block_tile, ldsc, m, n, warp_row_idx, warp_col_idx,
        thread_linear_row_idx_in_warp, thread_linear_col_idx_in_warp);

    // Check register add result
    // if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0)
    // {
    //     for(size_t x_i{0}; x_i < BLOCK_TILE_SIZE_X; ++x_i)
    //     {
    //         for(size_t y_i{0}; y_i < BLOCK_TILE_SIZE_Y; ++y_i)
    //         {
    //             T sum{0};
    //             for(size_t k_i{0}; k_i < BLOCK_TILE_SIZE_K; ++k_i)
    //             {
    //                 sum += A[(blockIdx.x * BLOCK_TILE_SIZE_X + x_i) +
    //                     (blockIdx.y * BLOCK_TILE_SIZE_K + k_i) * lda] *
    //                     B[(blockIdx.y * BLOCK_TILE_SIZE_K + k_i) +
    //                     (y_i) * ldb];
    //             }
    //             if(sum != C_thread_block_tile[x_i + y_i * ldsc])
    //             {
    //                 printf("C reg add error x_i: %ld y_i: %ld sum: %f c_block_tile: %f\n", x_i, y_i, sum, C_thread_block_tile[x_i + y_i * ldsc]);
    //             }
    //         }
    //     }
    // }

    // Copy the results of blocks to global memeory.
    // copy_results_from_shared_to_global_memory<T, BLOCK_TILE_SIZE_X,
    //     BLOCK_TILE_SIZE_Y, NUM_THREADS_PER_BLOCK>
    //     (blockIdx.x, blockIdx.y,
    //     C_thread_block_tile, ldsc, C_work, ldc,
    //     warp_linear_idx, thread_linear_idx_in_warp);

}

template <typename T>
void launch_gemm_kernel_tall_and_skinny_03(size_t m, size_t n, size_t k, T const* alpha,
                            T const* A, size_t lda, T const* B, size_t ldb,
                            T const* beta, T* C, size_t ldc,
                            hipStream_t stream)
{
    constexpr unsigned int BLOCK_TILE_SIZE_X{64U};
    constexpr unsigned int BLOCK_TILE_SIZE_K{64U};
    constexpr unsigned int BLOCK_TILE_SIZE_Y{32U};  // == n

    if(m % BLOCK_TILE_SIZE_X != 0 || n % BLOCK_TILE_SIZE_Y != 0 ||
        k % BLOCK_TILE_SIZE_K != 0)
    {
        printf("Block constraint not satisfied\n");
        return;
    }

    constexpr unsigned int WARP_TILE_SIZE_X{32U};
    constexpr unsigned int WARP_TILE_SIZE_Y{32U};   // == n

    constexpr unsigned int NUM_THREADS_PER_WARP_X{8U};
    constexpr unsigned int NUM_THREADS_PER_WARP_Y{4U};
    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32U);

    constexpr unsigned int THREAD_TILE_SIZE_X{4U};
    constexpr unsigned int THREAD_TILE_SIZE_Y{8U};

    static_assert(WARP_TILE_SIZE_X % THREAD_TILE_SIZE_X == 0U);
    static_assert(WARP_TILE_SIZE_Y % THREAD_TILE_SIZE_Y == 0U);
    static_assert(WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0U);
    static_assert(WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0U);

    constexpr unsigned int NUM_WARPS_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    constexpr unsigned int NUM_WARPS_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};

    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0U);
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0U);

    // The skew size is used to avoid bank conflicts in shared memory.
    constexpr size_t BLOCK_TILE_SKEW_SIZE_A{0U};
    constexpr size_t BLOCK_TILE_SKEW_SIZE_B{1U};

    constexpr unsigned int NUM_THREADS_PER_BLOCK{NUM_WARPS_X * NUM_WARPS_Y * 32U};

    static_assert(BLOCK_TILE_SIZE_X % 32U == 0);
    static_assert(((BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_K) % NUM_THREADS_PER_BLOCK) == 0);
    constexpr unsigned int NUM_EACH_THREAD_LOAD_A{
        (BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_K) / NUM_THREADS_PER_BLOCK};
    static_assert((BLOCK_TILE_SIZE_K) % (NUM_EACH_THREAD_LOAD_A) == 0);

    static_assert(BLOCK_TILE_SIZE_K % 32U == 0);
    static_assert(((BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_Y) % NUM_THREADS_PER_BLOCK) == 0);
    constexpr unsigned int NUM_EACH_THREAD_LOAD_B{
        (BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_Y) / NUM_THREADS_PER_BLOCK};
    static_assert((BLOCK_TILE_SIZE_Y) % (NUM_EACH_THREAD_LOAD_B) == 0);

    dim3 const block_dim{NUM_THREADS_PER_BLOCK, 1U, 1U};
    dim3 const grid_dim{
        (static_cast<unsigned int>(m) + BLOCK_TILE_SIZE_X - 1U) /
            BLOCK_TILE_SIZE_X,
        (static_cast<unsigned int>(k) + BLOCK_TILE_SIZE_K - 1U) /
            BLOCK_TILE_SIZE_K,
        1U};

    // printf("Block Dim: %d Grid Dim.x: %d Grid Dim.y: %d\n", block_dim.x, grid_dim.x, grid_dim.y);

    gemm_tall_and_skinny<T, BLOCK_TILE_SIZE_X, BLOCK_TILE_SIZE_K, BLOCK_TILE_SIZE_Y,
                         BLOCK_TILE_SKEW_SIZE_A, BLOCK_TILE_SKEW_SIZE_B,
                         WARP_TILE_SIZE_X, WARP_TILE_SIZE_Y,
                         NUM_THREADS_PER_WARP_X, NUM_THREADS_PER_WARP_Y,
                         THREAD_TILE_SIZE_X, THREAD_TILE_SIZE_Y>
                         <<<grid_dim, block_dim, 0U, stream>>>
                         (m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

// Explicit instantiation.
template void launch_gemm_kernel_tall_and_skinny_03<float>(size_t m, size_t n, size_t k,
                                                           float const* alpha,
                                                           float const* A, size_t lda,
                                                           float const* B, size_t ldb,
                                                           float const* beta, float* C,
                                                           size_t ldc, hipStream_t stream);

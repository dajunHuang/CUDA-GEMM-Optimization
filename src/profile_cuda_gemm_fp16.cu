#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include "cuda_gemm.hpp"
#include "profile_utils.cuh"

int main()
{
    print_device_info();

    constexpr size_t num_repeats{1U};
    constexpr size_t num_warmups{1U};

    __half const fp16_tensor_core_abs_tol{__float2half(5.0e-2f)};
    double const fp16_tensor_core_rel_tol{1.0e-1f};

    constexpr size_t m{8192U};
    constexpr size_t k{8192U};
    constexpr size_t n{8192U};

    constexpr size_t lda{(m + 16U - 1U) / 16U * 16U};
    constexpr size_t ldb{(k + 16U - 1U) / 16U * 16U};
    constexpr size_t ldc{(m + 16U - 1U) / 16U * 16U};

    static_assert(lda >= m);
    static_assert(ldb >= k);
    static_assert(ldc >= m);

    std::cout << "Matrix Size: " << "M = " << m << " N = " << n << " K = " << k
              << std::endl;
    std::cout << "Matrix A: " << m << " x " << k
              << " Leading Dimension Size = " << lda << std::endl;
    std::cout << "Matrix B: " << k << " x " << n
              << " Leading Dimension Size = " << ldb << std::endl;
    std::cout << "Matrix C: " << m << " x " << n
              << " Leading Dimension Size = " << ldc << std::endl;
    std::cout << std::endl;

    std::vector<std::pair<
        std::string,
        std::function<void(size_t, size_t, size_t, __half const*, __half const*,
                           size_t, __half const*, size_t, __half const*,
                           __half*, size_t, hipStream_t)>>> const
        gemm_fp16_tensor_core_kernel_launch_functions{

        };

    for (auto const& gemm_fp16_tensor_core_kernel_launch_function :
         gemm_fp16_tensor_core_kernel_launch_functions)
    {
        std::cout << gemm_fp16_tensor_core_kernel_launch_function.first
                  << std::endl;
        std::pair<__half, __half> const gemm_kernel_profile_result{
            profile_gemm<__half>(
                m, n, k, lda, ldb, ldc,
                gemm_fp16_tensor_core_kernel_launch_function.second,
                fp16_tensor_core_abs_tol, fp16_tensor_core_rel_tol, num_repeats,
                num_warmups)};
        std::cout << std::endl;
    }

    return 0;
}